#include "hip/hip_runtime.h"
#include "catch2/matchers/catch_matchers.hpp"
#include "cuda_path_tracer/error.cuh"
#define protected public
#include "cuda_path_tracer/shapes/rotation.cuh"
#include <catch2/catch_test_macros.hpp>
#include <catch2/matchers/catch_matchers_floating_point.hpp>

const auto epsilon = 1e-6f;

__global__ void testRotatePoint(const Rotation *rotation, Vec3 *point,
                                Vec3 *result, bool inverse) {
  *result = rotation->rotate(*point, inverse);
}

// NOLINTBEGIN(cppcoreguidelines-avoid-do-while,cppcoreguidelines-avoid-magic-numbers)

TEST_CASE("Rotation Construction and Caching", "[rotation]") {

  SECTION("Zero rotation") {
    Rotation rot(Vec3(0.0f, 0.0f, 0.0f));
    Vec3 *d_point = nullptr;
    Vec3 *d_result = nullptr;
    Rotation *d_rotation = nullptr;

    CUDA_ERROR_CHECK(hipMallocManaged(&d_point, sizeof(Vec3)));
    CUDA_ERROR_CHECK(hipMallocManaged(&d_result, sizeof(Vec3)));
    CUDA_ERROR_CHECK(hipMallocManaged(&d_rotation, sizeof(Rotation)));

    *d_point = Vec3(1.0f, 0.0f, 0.0f);
    *d_rotation = rot;

    testRotatePoint<<<1, 1>>>(d_rotation, d_point, d_result, false);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    REQUIRE_THAT(d_result->x, Catch::Matchers::WithinAbs(1.0f, epsilon));
    REQUIRE_THAT(d_result->y, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->z, Catch::Matchers::WithinAbs(0.0f, epsilon));

    CUDA_ERROR_CHECK(hipFree(d_point));
    CUDA_ERROR_CHECK(hipFree(d_result));
    CUDA_ERROR_CHECK(hipFree(d_rotation));
  }

  SECTION("90 degree X rotation") {
    Rotation rot(Vec3(90.0f, 0.0f, 0.0f));
    Vec3 *d_point = nullptr;
    Vec3 *d_result = nullptr;
    Rotation *d_rotation = nullptr;

    CUDA_ERROR_CHECK(hipMallocManaged(&d_point, sizeof(Vec3)));
    CUDA_ERROR_CHECK(hipMallocManaged(&d_result, sizeof(Vec3)));
    CUDA_ERROR_CHECK(hipMallocManaged(&d_rotation, sizeof(Rotation)));

    *d_point = Vec3(0.0f, 1.0f, 0.0f);
    *d_rotation = rot;

    testRotatePoint<<<1, 1>>>(d_rotation, d_point, d_result, false);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    REQUIRE_THAT(d_result->x, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->y, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->z, Catch::Matchers::WithinAbs(1.0f, epsilon));

    CUDA_ERROR_CHECK(hipFree(d_point));
    CUDA_ERROR_CHECK(hipFree(d_result));
    CUDA_ERROR_CHECK(hipFree(d_rotation));
  }

  SECTION("Combined rotation") {
    Rotation rot(Vec3(90.0f, 90.0f, 90.0f));
    Vec3 point(0.0f, 1.0f, 0.0f);
    Vec3 *d_point = nullptr;
    Vec3 *d_result = nullptr;
    Rotation *d_rotation = nullptr;

    CUDA_ERROR_CHECK(hipMallocManaged(&d_point, sizeof(Vec3)));
    CUDA_ERROR_CHECK(hipMallocManaged(&d_result, sizeof(Vec3)));
    CUDA_ERROR_CHECK(hipMallocManaged(&d_rotation, sizeof(Rotation)));

    CUDA_ERROR_CHECK(
        hipMemcpy(d_point, &point, sizeof(Vec3), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(
        hipMemcpy(d_rotation, &rot, sizeof(Rotation), hipMemcpyHostToDevice));

    testRotatePoint<<<1, 1>>>(d_rotation, d_point, d_result, false);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    // Values for 90-degree rotation on all axes
    REQUIRE_THAT(d_result->x, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->y, Catch::Matchers::WithinAbs(1.0f, epsilon));
    REQUIRE_THAT(d_result->z, Catch::Matchers::WithinAbs(0.0f, epsilon));

    CUDA_ERROR_CHECK(hipFree(d_point));
    CUDA_ERROR_CHECK(hipFree(d_result));
    CUDA_ERROR_CHECK(hipFree(d_rotation));
  }
}

// FIXME: Inverse rotation is broken, maybe it's a problem with the code
TEST_CASE("Inverse Rotation Tests", "[rotation]") {
  Vec3 *d_point = nullptr;
  Vec3 *d_result = nullptr;
  Vec3 *d_final = nullptr;
  Rotation *d_rotation = nullptr;

  CUDA_ERROR_CHECK(hipMallocManaged(&d_point, sizeof(Vec3)));
  CUDA_ERROR_CHECK(hipMallocManaged(&d_result, sizeof(Vec3)));
  CUDA_ERROR_CHECK(hipMallocManaged(&d_final, sizeof(Vec3)));
  CUDA_ERROR_CHECK(hipMallocManaged(&d_rotation, sizeof(Rotation)));

  SECTION("Inverse X rotation 90 degrees") {
    Rotation rot(Vec3(90.0f, 0.0f, 0.0f));
    Vec3 point(0.0f, 1.0f, 0.0f);
    CUDA_ERROR_CHECK(
        hipMemcpy(d_point, &point, sizeof(Vec3), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(
        hipMemcpy(d_rotation, &rot, sizeof(Rotation), hipMemcpyHostToDevice));

    testRotatePoint<<<1, 1>>>(d_rotation, d_point, d_result, true);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    REQUIRE_THAT(d_result->x, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->y, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->z, Catch::Matchers::WithinAbs(-1.0f, epsilon));
  }

  SECTION("Rotation followed by inverse returns original point") {
    Rotation rot(Vec3(45.0f, 30.0f, 60.0f));
    Vec3 point(1.0f, 0.0f, 0.0f);
    CUDA_ERROR_CHECK(
        hipMemcpy(d_point, &point, sizeof(Vec3), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(
        hipMemcpy(d_rotation, &rot, sizeof(Rotation), hipMemcpyHostToDevice));

    // Forward rotation
    testRotatePoint<<<1, 1>>>(d_rotation, d_point, d_result, false);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    // Inverse rotation
    testRotatePoint<<<1, 1>>>(d_rotation, d_result, d_final, true);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    REQUIRE_THAT(d_final->x, Catch::Matchers::WithinAbs(1.0f, epsilon));
    REQUIRE_THAT(d_final->y, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_final->z, Catch::Matchers::WithinAbs(0.0f, epsilon));
  }

  SECTION("Inverse combined rotation") {
    Rotation rot(Vec3(90.0f, 90.0f, 90.0f));
    Vec3 point(0.0f, 1.0f, 0.0f);

    CUDA_ERROR_CHECK(
        hipMemcpy(d_point, &point, sizeof(Vec3), hipMemcpyHostToDevice));
    CUDA_ERROR_CHECK(
        hipMemcpy(d_rotation, &rot, sizeof(Rotation), hipMemcpyHostToDevice));

    testRotatePoint<<<1, 1>>>(d_rotation, d_point, d_result, true);
    CUDA_ERROR_CHECK(hipDeviceSynchronize());
    CUDA_ERROR_CHECK(hipGetLastError());

    REQUIRE_THAT(d_result->x, Catch::Matchers::WithinAbs(1.0f, epsilon));
    REQUIRE_THAT(d_result->y, Catch::Matchers::WithinAbs(0.0f, epsilon));
    REQUIRE_THAT(d_result->z, Catch::Matchers::WithinAbs(0.0f, epsilon));
  }

  CUDA_ERROR_CHECK(hipFree(d_point));
  CUDA_ERROR_CHECK(hipFree(d_result));
  CUDA_ERROR_CHECK(hipFree(d_final));
  CUDA_ERROR_CHECK(hipFree(d_rotation));
}

// NOLINTEND(cppcoreguidelines-avoid-do-while,cppcoreguidelines-avoid-magic-numbers)