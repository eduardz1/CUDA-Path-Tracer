#include "hip/hip_runtime.h"
/**
 * @file render.cu
 * @author Eduard Occhipinti (occhipinti.eduard@icloud.com)
 * @brief Implementation file for render.cuh, which contains the kernel for
 * rendering the image
 * @version 0.1
 * @date 2024-10-26
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "cuda_path_tracer/ray.cuh"
#include "cuda_path_tracer/render.cuh"

#include <vector_functions.h>

__global__ void renderImage(int width, int height, uchar4 *image) {
  auto x = blockIdx.x * blockDim.x + threadIdx.x;
  auto y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) {
    return;
  }

  auto index = y * width + x;

  Ray const r(Vec3(0, 0, 0), Vec3(0, 0, 1));

  // Save the pixel for the R G B and Alpha values
  image[index] = make_uchar4(UCHAR_MAX, 0, 0, UCHAR_MAX); // TODO: Make it query a ray
}
