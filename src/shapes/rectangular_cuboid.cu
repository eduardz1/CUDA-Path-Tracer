#include "hip/hip_runtime.h"
#include "cuda_path_tracer/shapes/rectangular_cuboid.cuh"
#include "cuda_path_tracer/vec3.cuh"
#include <array>

__host__ RectangularCuboid::RectangularCuboid(const Vec3 &a, const Vec3 &b)
    : a(a), b(b) {
  const auto min =
      Vec3(std::fmin(a.x, b.x), std::fmin(a.y, b.y), std::fmin(a.z, b.z));
  const auto max =
      Vec3(std::fmax(a.x, b.x), std::fmax(a.y, b.y), std::fmax(a.z, b.z));

  const auto dx = Vec3(max.x - min.x, 0, 0);
  const auto dy = Vec3(0, max.y - min.y, 0);
  const auto dz = Vec3(0, 0, max.z - min.z);

  faces.left = Parallelogram(min, dz, dy);
  faces.bottom = Parallelogram(min, dx, dz);
  faces.front = Parallelogram({min.x, min.y, max.z}, dx, dy);
  faces.right = Parallelogram({max.x, min.y, max.z}, -dz, dy);
  faces.back = Parallelogram({max.x, min.y, min.z}, -dx, dy);
  faces.top = Parallelogram({min.x, max.y, max.z}, dx, -dz);
};

__host__ auto
RectangularCuboid::rotate(const Vec3 &angles) -> RectangularCuboid & {
  this->rotation += Rotation(angles);
  return *this;
};
__host__ auto
RectangularCuboid::translate(const Vec3 &translation) -> RectangularCuboid & {
  this->translation += {-translation.x, translation.y, translation.z};
  return *this;
};

__device__ auto RectangularCuboid::hit(const Ray &r, const float hit_t_min,
                                       const float hit_t_max,
                                       HitInfo &hi) const -> bool {
  const auto origin = rotation.rotatePoint(r.getOrigin(), true);
  const auto direction = rotation.rotatePoint(r.getDirection(), true);
  const Ray rotated_ray = {origin - this->translation, direction};

  HitInfo temp_hi;
  bool hit_any = false;
  float closest_t = hit_t_max;

  std::array<const Parallelogram *, 6> faces_arr{// NOLINT
                                                 &faces.left,  &faces.bottom,
                                                 &faces.front, &faces.right,
                                                 &faces.back,  &faces.top};

  for (auto &i : faces_arr) {
    if (i->hit(rotated_ray, hit_t_min, closest_t, temp_hi)) {
      hit_any = true;
      closest_t = temp_hi.getTime();
      hi = temp_hi;
    }
  }

  if (!hit_any) {
    return false;
  }

  hi.setPoint(rotation.rotatePoint(hi.getPoint(), false) + this->translation);
  hi.setNormal(rotation.rotatePoint(hi.getNormal(), false));

  return true;
}
