#include "hip/hip_runtime.h"
#include "cuda_path_tracer/hit_info.cuh"
#include "cuda_path_tracer/shapes/parallelogram.cuh"
#include "cuda_path_tracer/vec3.cuh"

__host__ Parallelogram::Parallelogram() = default;
__host__ Parallelogram::Parallelogram(const Vec3 &origin, const Vec3 &u,
                                      const Vec3 &v, const Material &material)
    : origin(origin), u(u), v(v), material(material) {
  const auto n = cross(u, v);
  normal = makeUnitVector(n);
  area = dot(normal, origin); // NOLINT
  w = n / dot(n, n);
};

__device__ auto Parallelogram::hit(const Ray &r, const float hit_t_min,
                                   const float hit_t_max, HitInfo &hi) const
    -> bool {
  const auto denominator = dot(normal, r.getDirection());

  if (fabs(denominator) < 1e-6) { // NOLINT ray is parallel to the plane
    return false;
  }

  const auto t = (area - dot(normal, r.getOrigin())) / denominator;
  if (t < hit_t_min || t > hit_t_max) {
    return false;
  }

  const auto point = r.at(t);

  const auto p = point - origin;
  const auto alpha = dot(w, cross(p, v));
  const auto beta = dot(w, cross(u, p));

  if (alpha < 0 || alpha > 1 || beta < 0 || beta > 1) {
    return false;
  }

  hi.time = t;
  hi.point = point;
  hi.setNormal(r, normal);
  hi.material = material;

  return true;
}