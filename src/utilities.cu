#include "cuda_path_tracer/error.cuh"
#include "cuda_path_tracer/utilities.cuh"

StreamGuard::StreamGuard(){CUDA_ERROR_CHECK(hipStreamCreate(&stream))};

StreamGuard::~StreamGuard() {
  if (stream != nullptr) {
    CUDA_ERROR_CHECK(hipStreamDestroy(stream))
  }
};

[[nodiscard]] auto StreamGuard::get() const -> hipStream_t { return stream; }
StreamGuard::operator hipStream_t() const { return stream; }

auto StreamGuard::operator=(StreamGuard &&other) noexcept -> StreamGuard & {
  if (this != &other) {
    if (stream != nullptr) {
      CUDA_ERROR_CHECK(hipStreamDestroy(stream));
    }
    stream = other.stream;
    other.stream = nullptr;
  }
  return *this;
}
StreamGuard::StreamGuard(StreamGuard &&other) noexcept : stream(other.stream) {
  other.stream = nullptr;
}
