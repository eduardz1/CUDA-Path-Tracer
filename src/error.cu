#include "cuda_path_tracer/error.cuh"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

void cudaAssert(const hipError_t code, const char *file, const int line) {
  if (code == hipSuccess) {
    return;
  }

  std::cerr << "CUDA Error (" << code << "): " << hipGetErrorString(code)
            << " " << file << " " << line << '\n';

  std::quick_exit(code);
}