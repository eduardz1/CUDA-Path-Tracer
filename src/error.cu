#include "cuda_path_tracer/error.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

void cudaAssert(hipError_t code, const char *file, int line) {
  if (code == hipSuccess)
    return;

  std::cerr << "CUDA Error: " << hipGetErrorString(code) << " " << file << " "
            << line << std::endl;

  exit(code);
}