#include "cuda_path_tracer/error.cuh"
#include <cstdio>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

inline void cudaAssert(hipError_t code, const char *file, int line) {
  if (code == hipSuccess)
    return;

  fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file,
          line);
  exit(code);
}