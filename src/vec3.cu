#include "hip/hip_runtime.h"
#include "cuda_path_tracer/vec3.cuh"
#include <cmath>

__host__ __device__ auto Vec3::operator-() const -> Vec3 {
  return {-x, -y, -z};
}
__host__ __device__ auto Vec3::operator==(const Vec3 &other) const -> bool {
  return x == other.x && y == other.y && z == other.z;
}
__host__ __device__ auto Vec3::operator+=(const Vec3 &other) -> Vec3 & {
  x += other.x;
  y += other.y;
  z += other.z;

  return *this;
}
__device__ Vec3::operator float4() const { return make_float4(x, y, z, 1.0F); }

__host__ __device__ auto Vec3::getLengthSquared() const -> float {
  return x * x + y * y + z * z;
}

__host__ __device__ auto Vec3::getLength() const -> float {
  return sqrtf(getLengthSquared());
}

__host__ auto operator<<(std::ostream &os, const Vec3 &v) -> std::ostream & {
  os << "(" << v.x << ", " << v.y << ", " << v.z << ")";
  return os;
}
__host__ __device__ auto operator+(const Vec3 &v1, const Vec3 &v2) -> Vec3 {
  return {v1.x + v2.x, v1.y + v2.y, v1.z + v2.z};
}
__host__ __device__ auto operator-(const Vec3 &v1, const Vec3 &v2) -> Vec3 {
  return {v1.x - v2.x, v1.y - v2.y, v1.z - v2.z};
}
__host__ __device__ auto operator*(const Vec3 &v, const float t) -> Vec3 {
  return {t * v.x, t * v.y, t * v.z};
}
__host__ __device__ auto operator*(const Vec3 &v1, const Vec3 &v2) -> Vec3 {
  return {v1.x * v2.x, v1.y * v2.y, v1.z * v2.z};
}
__host__ __device__ auto operator/(const Vec3 &v, float t) -> Vec3 {
  return {v.x / t, v.y / t, v.z / t};
}

__device__ auto randomVector(hiprandStatePhilox4_32_10_t &state) -> Vec3 {
  return Vec3{hiprand_uniform(&state), hiprand_uniform(&state),
              hiprand_uniform(&state)};
}

__host__ __device__ auto makeUnitVector(const Vec3 &v) -> Vec3 {
  return v / v.getLength();
}

__host__ __device__ auto dot(const Vec3 &v1, const Vec3 &v2) -> float {
  return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__host__ __device__ auto cross(const Vec3 &v1, const Vec3 &v2) -> Vec3 {
  return {v1.y * v2.z - v1.z * v2.y, v1.z * v2.x - v1.x * v2.z,
          v1.x * v2.y - v1.y * v2.x};
}

__device__ auto vectorOnHemisphere(const Vec3 &v,
                                   hiprandStatePhilox4_32_10_t &state) -> Vec3 {
  Vec3 randomUnit = makeUnitVector(randomVector(state));

  if (dot(randomUnit, v) > 0.0) {
    return randomUnit;
  }
  return -randomUnit;
}

__device__ auto roundScatterDirection(const Vec3 &direction,
                                      const Vec3 &normal) -> Vec3 {
  const auto s = 1e-8F;
  if (std::fabs(direction.x) < s && std::fabs(direction.y) < s &&
      std::fabs(direction.z) < s) {
    return normal;
  }
  return direction;
}

__device__ auto reflect(const Vec3 &v, const Vec3 &n) -> Vec3 {
  return v - 2 * dot(v, n) * n;
}

__device__ auto refract(const Vec3 &v, const Vec3 &n,
                        float eta_component) -> Vec3 {
  const auto cos_theta = static_cast<float>(std::fmin(dot(-v, n), 1.0));
  const Vec3 r_perp = eta_component * (v + cos_theta * n);
  const Vec3 r_par =
      -sqrtf(std::fabs(1.0F - r_perp.getLengthSquared())) * n;
  return r_perp + r_par;
}