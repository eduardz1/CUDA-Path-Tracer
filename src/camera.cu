#include "hip/hip_runtime.h"
#include <climits>
#include <cstdint>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_path_tracer/camera.cuh"
#include "cuda_path_tracer/error.cuh"
#include "cuda_path_tracer/hit_info.cuh"
#include "cuda_path_tracer/image.cuh"
#include "cuda_path_tracer/ray.cuh"
#include "cuda_path_tracer/shape.cuh"
#include "cuda_path_tracer/vec3.cuh"

namespace {
constexpr unsigned long long SEED = 0xba0bab;
constexpr dim3 BLOCK_SIZE(16, 16);

__device__ auto getRay(const Vec3 origin, const Vec3 pixel00, const Vec3 deltaU,
                       const Vec3 deltaV, const uint16_t x, const uint16_t y,
                       hiprandState &state) -> Ray {
  // We sample an area of "half pixel" around the pixel centers
  auto offset =
      Vec3{hiprand_uniform(&state) - 0.5f, hiprand_uniform(&state) - 0.5f, 0};
  auto sample = pixel00 + ((float(x) + offset.getX()) * deltaU) +
                ((float(y) + offset.getY()) * deltaV);

  // auto center = pixel00 + deltaU * x + deltaV * y;
  // auto direction = sample - center;

  // return {origin, direction};
  return {origin, sample - origin};
}

/**
 * @brief Saves the closest hit information in the HitInfo struct from the given
 * ray and shapes. Returns true if a hit was found, false otherwise.
 *
 * @param ray Ray to check for hits
 * @param shapes Array of shapes to check for hits
 * @param num_shapes Number of shapes in the array
 * @param hi HitInfo struct to save the hit information
 * @return bool true if a hit was found, false otherwise
 */
__device__ auto hitShapes(const Ray &ray, const Shape *shapes,
                          const size_t num_shapes, HitInfo &hi) -> bool {
  auto tmp = HitInfo();
  auto closest = RAY_T_MAX;
  auto hit_anything = false;
  for (size_t i = 0; i < num_shapes; i++) {
    const bool hit = cuda::std::visit(
        [&ray, &tmp, closest](const auto &shape) {
          return shape.hit(ray, RAY_T_MIN, closest, tmp);
        },
        shapes[i]);

    if (hit) {
      hit_anything = true;
      closest = tmp.getTime();
      hi = tmp;
    }
  }
  return hit_anything;
}

__device__ auto getColor(const Ray &ray, const Shape *shapes,
                         const size_t num_shapes) -> Vec3 {
  auto hi = HitInfo();
  const bool hit = hitShapes(ray, shapes, num_shapes, hi);

  if (hit) {
    return 0.5f * (hi.getNormal() + 1.0f);
  }

  auto unit_direction = makeUnitVector(ray.getDirection());
  auto t = 0.5f * (unit_direction.getY() + 1.0f);
  return (1.0f - t) * Vec3{1.0f} + t * Vec3{0.5f, 0.7f, 1.0f};
}

/**
 * @brief Kernel for rendering the image, works by calculating the pixel index
 * in the image, computing the Ray that goes from the camera's origin to the
 * pixel center, querying it for a color and then saving this color value in the
 * image buffer.
 *
 * @param width Width of the image
 * @param height Height of the image
 * @param image Image to render
 * @param origin Camera's origin
 * @param pixel00 Pixel at the top left corner of the image
 * @param deltaU Horizontal vector of the distance between each pixel center
 * @param deltaV Vertical vector of the distance between each pixel center
 * @param shapes Array of shapes to check for hits
 * @param num_shapes Number of shapes in the array
 * @param states Random number generator states for each pixel
 * @param num_samples_ppx Number of samples for each pixel
 */
__global__ void renderImage(const uint16_t width, const uint16_t height,
                            uchar4 *image, const Vec3 origin,
                            const Vec3 pixel00, const Vec3 deltaU,
                            const Vec3 deltaV, const Shape *shapes,
                            const size_t num_shapes, hiprandState *states,
                            const uint8_t num_samples_ppx) {
  const auto x = blockIdx.x * blockDim.x + threadIdx.x;
  const auto y = blockIdx.y * blockDim.y + threadIdx.y;

  if (x >= width || y >= height) {
    return;
  }

  const auto index = y * width + x;
  hiprand_init(SEED, index, 0, &states[index]);

  hiprandState state = states[index];
  auto color = Vec3{};
  for (auto s = 0; s < num_samples_ppx; s++) {
    const auto ray = getRay(origin, pixel00, deltaU, deltaV, x, y, state);
    color += getColor(ray, shapes, num_shapes);
  }

  image[index] = convertColorTo8Bit(color / float(num_samples_ppx));
}
} // namespace

__host__ Camera::Camera() : origin() {}
__host__ Camera::Camera(const Vec3 &origin) : origin(origin) {}

__host__ void Camera::render(const std::shared_ptr<Scene> &scene,
                             uchar4 *image) {
  const auto width = scene->getWidth();
  const auto height = scene->getHeight();
  hiprandState *states;
  CUDA_ERROR_CHECK(
      hipMalloc((void **)&states, width * height * sizeof(hiprandState)));

  const std::vector<Shape> &h_shapes = scene->getShapes();
  const size_t num_shapes = h_shapes.size();
  Shape *d_shapes;
  CUDA_ERROR_CHECK(hipMalloc((void **)&d_shapes, num_shapes * sizeof(Shape)));
  CUDA_ERROR_CHECK(hipMemcpy(d_shapes, h_shapes.data(),
                              num_shapes * sizeof(Sphere),
                              hipMemcpyHostToDevice));

  viewportWidth = (float(width) / float(height)) * viewportHeight;

  auto viewportU = Vec3(viewportWidth, 0, 0);
  auto viewportV = Vec3(0, -viewportHeight, 0);

  deltaU = viewportU / float(width);
  deltaV = viewportV / float(height);

  pixel00 =
      (origin - viewportU / 2 - viewportV / 2 + origin) + (deltaU + deltaV) / 2;

  uchar4 *image_device;

  const auto size = static_cast<long>(width) * height * sizeof(uchar4);

  CUDA_ERROR_CHECK(hipMalloc((void **)&image_device, size));

  dim3 grid((width + BLOCK_SIZE.x - 1) / BLOCK_SIZE.x,
            (height + BLOCK_SIZE.y - 1) / BLOCK_SIZE.y);

  renderImage<<<grid, BLOCK_SIZE>>>(width, height, image_device, origin,
                                    pixel00, deltaU, deltaV, d_shapes,
                                    num_shapes, states, this->num_samples_ppx);
  hipDeviceSynchronize();
  CUDA_ERROR_CHECK(hipGetLastError());

  CUDA_ERROR_CHECK(
      hipMemcpy(image, image_device, size, hipMemcpyDeviceToHost));

  hipFree(d_shapes);
  CUDA_ERROR_CHECK(hipGetLastError());
}
