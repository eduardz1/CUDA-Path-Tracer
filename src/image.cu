#include "hip/hip_runtime.h"
/**
 * @file image.cpp
 * @author Eduard Occhipinti (occhipinti.eduard@icloud.com)
 * @brief Implementation file for image.hh, which contains the functions to save
 * an image as a PPM file
 * @version 0.1
 * @date 2024-10-26
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "cuda_path_tracer/image.cuh"
#include <algorithm>
#include <fstream>
#include <vector_functions.h>

__host__ void saveImageAsPPM(const std::string &filename, const uint16_t width,
                             const uint16_t height,
                             const thrust::host_vector<uchar4> &image) {
  std::ofstream file(filename);

  file << "P3\n";
  file << width << " " << height << "\n";
  file << UCHAR_MAX << "\n";

  for (int i = 0; i < width * height; i++) {
    file << +image[i].x << " " << +image[i].y << " " << +image[i].z << "\n";
  }

  file.close();
}

__device__ auto convertColorTo8Bit(const float4 color) -> uchar4 {
  return make_uchar4(
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.x, 0.0f, 1.0f)),
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.y, 0.0f, 1.0f)),
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.z, 0.0f, 1.0f)),
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.w, 0.0f, 1.0f)));
}
