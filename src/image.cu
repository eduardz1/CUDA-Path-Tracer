#include "hip/hip_runtime.h"
/**
 * @file image.cpp
 * @author Eduard Occhipinti (occhipinti.eduard@icloud.com)
 * @brief Implementation file for image.hh, which contains the functions to save
 * an image as a PPM file
 * @version 0.1
 * @date 2024-10-26
 *
 * @copyright Copyright (c) 2024
 *
 */

#include "cuda_path_tracer/image.cuh"
#include <algorithm>
#include <fstream>
#include <vector>
#include <vector_functions.h>

__host__ void saveImageAsPPM(const char *filename, const int width,
                             const int height,
                             const std::vector<uchar4> &image) {
  std::ofstream file(filename);

  file << "P3\n";
  file << width << " " << height << "\n";
  file << UCHAR_MAX << "\n";

  for (int i = 0; i < width * height; i++) {
    file << +image[i].x << " " << +image[i].y << " " << +image[i].z << "\n";
  }

  file.close();
}

__device__ auto convertColorTo8Bit(const float4 color) -> uchar4 {
  return make_uchar4(
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.x, 0.0f, 1.0f)),
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.y, 0.0f, 1.0f)),
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.z, 0.0f, 1.0f)),
      static_cast<unsigned char>(static_cast<float>(UCHAR_MAX) *
                                 std::clamp(color.w, 0.0f, 1.0f)));
}
