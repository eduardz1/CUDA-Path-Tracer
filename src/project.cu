#include "hip/hip_runtime.h"
#include "cuda_path_tracer/color.cuh"
#include "cuda_path_tracer/project.cuh"
#include <fstream>
#include <nlohmann/json.hpp>

namespace {
// Compile time string hashing
constexpr auto hash_string(std::string_view str) -> uint64_t {
  uint64_t hash = 14695981039346656037ULL; // NOLINT
  for (char c : str) {
    hash ^= static_cast<uint64_t>(c);
    hash *= 1099511628211ULL; // NOLINT
  }
  return hash;
}

constexpr auto operator""_hash(const char *str, size_t /*unused*/) -> uint64_t {
  return hash_string(str);
}

__host__ auto parseColor(const nlohmann::json &j) -> Vec3 {
  // If it's an array, parse as RGB values
  if (j.is_array()) {
    return Vec3{j[0], j[1], j[2]};
  }

  // If it's a string, parse as color name
  const auto colorName = j.get<std::string>();

  switch (hash_string(colorName)) {
  case "white"_hash: return Colors::White;
  case "black"_hash: return Colors::Black;
  case "red"_hash: return Colors::Red;
  case "green"_hash: return Colors::Green;
  case "blue"_hash: return Colors::Blue;
  case "yellow"_hash: return Colors::Yellow;
  case "cyan"_hash: return Colors::Cyan;
  case "magenta"_hash: return Colors::Magenta;

  case "rosewater"_hash: return Catpuccin::Latte::Rosewater;
  case "flamingo"_hash: return Catpuccin::Latte::Flamingo;
  case "pink"_hash: return Catpuccin::Latte::Pink;
  case "mauve"_hash: return Catpuccin::Latte::Mauve;
  case "latte-red"_hash: return Catpuccin::Latte::Red;
  case "maroon"_hash: return Catpuccin::Latte::Maroon;
  case "peach"_hash: return Catpuccin::Latte::Peach;
  case "latte-yellow"_hash: return Catpuccin::Latte::Yellow;
  case "latte-green"_hash: return Catpuccin::Latte::Green;
  case "teal"_hash: return Catpuccin::Latte::Teal;
  case "sky"_hash: return Catpuccin::Latte::Sky;
  case "sapphire"_hash: return Catpuccin::Latte::Sapphire;
  case "latte-blue"_hash: return Catpuccin::Latte::Blue;
  case "lavander"_hash: return Catpuccin::Latte::Lavander;
  case "text"_hash: return Catpuccin::Latte::Text;
  case "base"_hash: return Catpuccin::Latte::Base;
  case "mantle"_hash: return Catpuccin::Latte::Mantle;
  case "crust"_hash: return Catpuccin::Latte::Crust;

  default: throw std::runtime_error("Unknown color name: " + colorName);
  }
}

__host__ auto parseVec3(const nlohmann::json &j) -> Vec3 {
  return Vec3{j[0], j[1], j[2]};
}

__host__ auto parseMaterial(const nlohmann::json &j) -> Material {
  const auto type = j["type"].get<std::string>();

  switch (hash_string(type)) {
  case "lambertian"_hash: return Lambertian(parseColor(j["color"]));
  case "dielectric"_hash: return Dielectric(j["refraction_index"].get<float>());
  case "metal"_hash:
    return Metal(parseColor(j["color"]), j["fuzz"].get<float>());

  default: throw std::runtime_error("Unknown material type: " + type);
  }
}

__host__ auto parseShape(const nlohmann::json &j) -> Shape {
  const auto type = j["type"].get<std::string>();

  switch (hash_string(type)) {
  case "sphere"_hash:
    return Sphere{parseVec3(j["center"]), j["radius"].get<float>(),
                  parseMaterial(j["material"])};
  case "rectangular_cuboid"_hash: {
    auto r = RectangularCuboid{
        parseVec3(j["vertices"][0]),
        parseVec3(j["vertices"][1]),
    };

    if (j.contains("rotation")) {
      r = r.rotate(parseVec3(j["rotation"]));
    }
    if (j.contains("translation")) {
      r = r.translate(parseVec3(j["translation"]));
    }

    return r;
  }
  case "parallelogram"_hash:
    return Parallelogram{parseVec3(j["origin"]), parseVec3(j["u"]),
                         parseVec3(j["v"])};

  default: throw std::runtime_error("Unknown shape type: " + type);
  }
}
} // namespace

__host__ auto Project::load(const std::string &filename)
    -> std::shared_ptr<Project> {
  std::ifstream file(filename);
  if (!file.is_open()) {
    throw std::runtime_error("Could not open file: " + filename);
  }

  nlohmann::json j;
  file >> j;

  auto project = std::make_shared<Project>();
  project->name = j["name"].get<std::string>() + ".ppm";

  // Parse image dimensions
  const auto width = j["image"]["width"].get<uint16_t>();
  const auto height = j["image"]["height"].get<uint16_t>();

  // Parse shapes
  thrust::device_vector<Shape> shapes;
  for (const auto &shape : j["shapes"]) {
    shapes.push_back(parseShape(shape));
  }

  // Create scene
  project->scene = std::make_shared<Scene>(width, height, shapes);

  // Parse camera settings
  const auto &cam = j["camera"];
  auto camera_builder = CameraBuilder();

  if (cam.contains("origin")) {
    camera_builder.origin(parseVec3(cam["origin"]));
  }
  if (cam.contains("lookAt")) {
    camera_builder.lookAt(parseVec3(cam["lookAt"]));
  }
  if (cam.contains("up")) {
    camera_builder.up(parseVec3(cam["up"]));
  }
  if (cam.contains("verticalFov")) {
    camera_builder.verticalFov(cam["verticalFov"].get<float>());
  }
  if (cam.contains("defocusAngle")) {
    camera_builder.defocusAngle(cam["defocusAngle"].get<float>());
  }
  if (cam.contains("focusDistance")) {
    camera_builder.focusDistance(cam["focusDistance"].get<float>());
  }
  if (cam.contains("background")) {
    camera_builder.background(parseColor(cam["background"]));
  }
  project->camera = std::make_shared<Camera<>>(camera_builder.build());

  return project;
}

__host__ auto Project::render() -> void {
  const auto width = scene->getWidth();
  const auto height = scene->getHeight();
  thrust::universal_host_pinned_vector<uchar4> image(
      static_cast<size_t>(width * height));

  camera->render(scene, image);

  saveImageAsPPM(this->name, width, height, image);
}
